#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>

#ifdef CRAYPAT
#include "pat_api.h"
#endif
#include "utils.h"
#include "update_halo.h"
//#include "apply_diffusion.h"
#include "apply_stencil.cuh"
#include "apply_stencil_cpu.h"

void apply_diffusion_hybrid(Storage3D<realType>& inField,
                            Storage3D<realType>& outField,
                            Storage3D<realType>& buffer,
                            realType const alpha,
                            unsigned const numIter,
                            double const z_slices_on_cpu) {
  // Utils
  std::size_t const xSize = inField.xSize();
  std::size_t const ySize = inField.ySize();
  std::size_t const xMin = inField.xMin();
  std::size_t const yMin = inField.yMin();
  std::size_t const xMax = inField.xMax();
  std::size_t const yMax = inField.yMax();
  std::size_t const zMin = inField.zMin();
  std::size_t const zMax = inField.zMax();
  std::size_t const zMax_gpu = zMax - z_slices_on_cpu;
  std::size_t const size = sizeof(realType) * xSize * ySize * zMax_gpu;

  // Allocate space on device memory and copy data from host
  realType *infield, *outfield;
  //hipInit(0);
  hipMalloc((void **)&infield, size);
  hipMalloc((void **)&outfield, size);
  hipMemcpy(infield, &inField(0, 0, 0), size, hipMemcpyHostToDevice);

  dim3 blockDim(8, 8, 1);
  dim3 gridDim((xMax + blockDim.x - 1) / blockDim.x,
               (yMax + blockDim.y - 1) / blockDim.y,
               (zMax_gpu + blockDim.z - 1) / blockDim.z);

  //hipEvent_t tic, toc;
  //hipEventCreate(&tic);
  //hipEventCreate(&toc);
  //hipEventRecord(tic);

  //Storage3D<realType> buffer(x, y, 1, halo);

  for(std::size_t iter = 0; iter < numIter; ++iter) {
    // GPU code (Control returns directly to the CPU)
    apply_stencil<<<gridDim, blockDim>>>(infield, outfield, xMin, xMax, xSize, yMin, yMax, ySize, zMax_gpu, alpha);
    // CPU code (Overlaps with GPU code)
    updateHalo(inField, zMax_gpu);
    apply_stencil_cpu(inField, outField, buffer, alpha, iter, numIter, zMax_gpu);
    // Synchronize
    hipDeviceSynchronize();
    if ( iter != numIter - 1 ) std::swap(infield, outfield);
  }

  //hipEventRecord(toc);
  //hipEventSynchronize(toc);
  //float telapsed = -1;
  //hipEventElapsedTime(&telapsed, tic, toc);
  //std::cout << "telapsed: " << telapsed << std::endl;
  //hipEventDestroy(tic);
  //hipEventDestroy(toc);

  // Copy result from device to host and free device memory
  hipMemcpy(&outField(0, 0, 0), outfield, size, hipMemcpyDeviceToHost);
  hipFree(infield);
  hipFree(outfield);
}

void reportTime(const Storage3D<realType>& storage, int nIter, double diff) {
  std::cout << "# ranks nx ny ny nz num_iter time\ndata = np.array( [ \\\n";
  int size = 1;
#pragma omp parallel
  {
#pragma omp master
    { size = omp_get_num_threads(); }
  }
  std::cout << "[ " << size << ", " << storage.xMax() - storage.xMin() << ", "
            << storage.yMax() - storage.yMin() << ", " << storage.zMax() << ", " << nIter << ", "
            << diff << "],\n";
  std::cout << "] )" << std::endl;
}

int main(int argc, char const* argv[]) {
#ifdef CRAYPAT
  PAT_record(PAT_STATE_OFF);
#endif
  int x = atoi(argv[2]);
  int y = atoi(argv[4]);
  int z = atoi(argv[6]);
  int iter = atoi(argv[8]);
  int z_slices_on_cpu = atoi(argv[10]);
  int nHalo = 2;
  assert(x > 0 && y > 0 && z > 0 && iter > 0);
  assert(z_slices_on_cpu >= 0 && z_slices_on_cpu <= z);
  std::cout << "Offloaded " << z_slices_on_cpu << " / " << z << " slices on the CPU" << std::endl;
  Storage3D<realType> input(x, y, z, nHalo);
  input.initialize();
  Storage3D<realType> output(x, y, z, nHalo);
  output.initialize();

  realType alpha = 1. / 32.;

  std::ofstream fout;
  fout.open("in_field.dat", std::ios::binary | std::ofstream::trunc);
  input.writeFile(fout);
  fout.close();
#ifdef CRAYPAT
  PAT_record(PAT_STATE_ON);
#endif
  Storage3D<realType> buffer(x, y, 1, nHalo);
  // Synchronize the host and device so that the timings are accurate
  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  apply_diffusion_hybrid(input, output, buffer, alpha, iter, z_slices_on_cpu);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
#ifdef CRAYPAT
  PAT_record(PAT_STATE_OFF);
#endif
  updateHalo(output);
  fout.open("out_field.dat", std::ios::binary | std::ofstream::trunc);
  output.writeFile(fout);
  fout.close();

  auto diff = end - start;
  double timeDiff = std::chrono::duration<double, std::milli>(diff).count() / 1000.;
  reportTime(output, iter, timeDiff);

  return 0;
}
