#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <fstream>
#include <iostream>
#include <omp.h>
#include <hip/hip_runtime.h>

#ifdef CRAYPAT
#include "pat_api.h"
#endif
#include "utils.h"
#include "update_halo.h"
//#include "apply_diffusion.h"
#include "apply_stencil.cuh"

void apply_diffusion_gpu(Storage3D<realType>& inField,
                         Storage3D<realType>& outField,
                         realType const alpha,
                         unsigned const numIter) {
  // Utils
  std::size_t const xSize = inField.xSize();
  std::size_t const ySize = inField.ySize();
  std::size_t const xMin = inField.xMin();
  std::size_t const yMin = inField.yMin();
  std::size_t const xMax = inField.xMax();
  std::size_t const yMax = inField.yMax();
  std::size_t const zMin = inField.zMin();
  std::size_t const zMax = inField.zMax();
  std::size_t const size = sizeof(realType) * xSize * ySize * zMax;

  // Allocate space on device memory and copy data from host
  realType *infield, *outfield;
  //hipInit(0);
  hipMalloc((void **)&infield, size);
  hipMalloc((void **)&outfield, size);
  hipMemcpy(infield, &inField(0, 0, 0), size, hipMemcpyHostToDevice);

  dim3 blockDim(8, 8, 1);
  dim3 gridDim((xMax + blockDim.x - 1) / blockDim.x,
               (yMax + blockDim.y - 1) / blockDim.y,
               (zMax + blockDim.z - 1) / blockDim.z);

  for(std::size_t iter = 0; iter < numIter; ++iter) {
    apply_stencil<<<gridDim, blockDim>>>(infield, outfield, xMin, xMax, xSize, yMin, yMax, ySize, zMax, alpha);
    hipDeviceSynchronize();
    if ( iter != numIter - 1 ) std::swap(infield, outfield);
  }

  // Copy result from device to host and free device memory
  hipMemcpy(&outField(0, 0, 0), outfield, size, hipMemcpyDeviceToHost);
  hipFree(infield);
  hipFree(outfield);
}

void reportTime(const Storage3D<realType>& storage, int nIter, double diff) {
  std::cout << "# ranks nx ny ny nz num_iter time\ndata = np.array( [ \\\n";
  int size = 1;
#pragma omp parallel
  {
#pragma omp master
    { size = omp_get_num_threads(); }
  }
  std::cout << "[ " << size << ", " << storage.xMax() - storage.xMin() << ", "
            << storage.yMax() - storage.yMin() << ", " << storage.zMax() << ", " << nIter << ", "
            << diff << "],\n";
  std::cout << "] )" << std::endl;
}

int main(int argc, char const* argv[]) {
#ifdef CRAYPAT
  PAT_record(PAT_STATE_OFF);
#endif
  int x = atoi(argv[2]);
  int y = atoi(argv[4]);
  int z = atoi(argv[6]);
  int iter = atoi(argv[8]);
  int nHalo = 2;
  assert(x > 0 && y > 0 && z > 0 && iter > 0);
  Storage3D<realType> input(x, y, z, nHalo);
  input.initialize();
  Storage3D<realType> output(x, y, z, nHalo);
  output.initialize();

  realType alpha = 1. / 32.;

  std::ofstream fout;
  fout.open("in_field.dat", std::ios::binary | std::ofstream::trunc);
  input.writeFile(fout);
  fout.close();
#ifdef CRAYPAT
  PAT_record(PAT_STATE_ON);
#endif
  // Synchronize the host and device so that the timings are accurate
  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  apply_diffusion_gpu(input, output, alpha, iter);

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
#ifdef CRAYPAT
  PAT_record(PAT_STATE_OFF);
#endif
  updateHalo(output);
  fout.open("out_field.dat", std::ios::binary | std::ofstream::trunc);
  output.writeFile(fout);
  fout.close();

  auto diff = end - start;
  double timeDiff = std::chrono::duration<double, std::milli>(diff).count() / 1000.;
  reportTime(output, iter, timeDiff);

  return 0;
}
